#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include "kernel.hpp"

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
#undef  assert
#define assert(arg)
#endif

__constant__ float* c_sf_e;
__constant__ float* c_sf_d;
__constant__ int c_sf_ns;
__constant__ float3 c_corner0;
__constant__ float3 c_corner1;
__constant__ float3 c_num_probes;
__constant__ float c_granularity_inverse;
__constant__ float* c_maps[sf_n];
__constant__ int c_num_generations;

extern __shared__ float shared[];

__device__  __noinline__// __forceinline__
bool evaluate(const float* x, float* e, float* g, float* a, float* q, float* c, float* d, float* f, float* t, const float e_upper_bound)
{
	return true;
}

__global__
//__launch_bounds__(maxThreadsPerBlock, minBlocksPerMultiprocessor)
void bfgs(float* __restrict__ s0e, float* __restrict__ s1e, float* __restrict__ s2e, const float* lig, const int nv, const int nf, const int na, const int np, const int seed)
{
	float h, s, c;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// Load constants and lig into shared memory.
	__syncthreads();
	sincosf(h, &s, &c);
}

kernel::kernel(const float* h_sf_e, const float* h_sf_d, const int h_sf_ns, const int h_sf_ne, const float* h_corner0, const float* h_corner1, const float* h_num_probes, const float h_granularity_inverse, const int num_mc_tasks, const int num_generations) : num_mc_tasks(num_mc_tasks), num_generations(num_generations)
{
	// Initialize scoring function.
	hipMalloc(&d_sf_e, h_sf_ne);
	hipMalloc(&d_sf_d, h_sf_ne);
	hipMemcpy(d_sf_e, h_sf_e, sizeof(float) * h_sf_ne, hipMemcpyHostToDevice);
	hipMemcpy(d_sf_d, h_sf_d, sizeof(float) * h_sf_ne, hipMemcpyHostToDevice);
	assert(sizeof(c_sf_e)  == sizeof(d_sf_e));
	assert(sizeof(c_sf_d)  == sizeof(d_sf_d));
	assert(sizeof(c_sf_ns) == sizeof(d_sf_ns));
	hipMemcpyToSymbol(HIP_SYMBOL(c_sf_e),  &d_sf_e,  sizeof(c_sf_e));
	hipMemcpyToSymbol(HIP_SYMBOL(c_sf_d),  &d_sf_d,  sizeof(c_sf_d));
	hipMemcpyToSymbol(HIP_SYMBOL(c_sf_ns), &h_sf_ns, sizeof(c_sf_ns));

	// Initialize receptor.
	assert(sizeof(c_corner0) == sizeof(float) * 3);
	assert(sizeof(c_corner1) == sizeof(float) * 3);
	assert(sizeof(c_num_probes) == sizeof(float) * 3);
	assert(sizeof(c_granularity_inverse) == sizeof(h_granularity_inverse));
	hipMemcpyToSymbol(HIP_SYMBOL(c_corner0), h_corner0, sizeof(c_corner0));
	hipMemcpyToSymbol(HIP_SYMBOL(c_corner1), h_corner1, sizeof(c_corner1));
	hipMemcpyToSymbol(HIP_SYMBOL(c_num_probes), h_num_probes, sizeof(c_num_probes));
	hipMemcpyToSymbol(HIP_SYMBOL(c_granularity_inverse), &h_granularity_inverse, sizeof(c_granularity_inverse));
	assert(sizeof(d_maps) == sizeof(float*) * sf_n);
	memset(d_maps, 0, sizeof(d_maps));
}

void kernel::update(const vector<vector<float> > h_maps, const size_t map_bytes, const vector<size_t>& xs)
{
	for (int i = 0; i < xs.size(); ++i)
	{
		const size_t t = xs[i];
		float* d_m;
		hipMalloc(&d_m, map_bytes);
		hipMemcpy(d_m, &h_maps[t].front(), map_bytes, hipMemcpyHostToDevice);
		d_maps[t] = d_m;
	}
	assert(sizeof(c_maps) == sizeof(d_maps));
	hipMemcpyToSymbol(HIP_SYMBOL(c_maps), d_maps, sizeof(c_maps));
}

void kernel::launch(vector<float>& h_ex, const int* h_lig, const int nv, const int nf, const int na, const int np, const size_t* seed)
{
	// Copy ligand content from host memory to device memory.
	const size_t lig_bytes = sizeof(int) * (11 * nf + nf - 1 + 4 * na + 3 * np);
	float* d_lig;
	hipMalloc(&d_lig, lig_bytes);
	hipMemcpy(d_lig, &h_lig, lig_bytes, hipMemcpyHostToDevice);

	// Allocate device memory for solutions.
	const size_t sln_bytes = sizeof(float) * (1 + (nv + 1) + nv + 3 * nf + 4 * nf + 3 * na + 3 * na + 3 * nf + 3 * nf) * num_mc_tasks;
	float* d_s0, d_s1, d_s2;
	hipMalloc(&d_s0, sln_bytes);
	hipMalloc(&d_s1, sln_bytes);
	hipMalloc(&d_s2, sln_bytes);

	// Invoke CUDA kernel.
	bfgs<<<num_mc_tasks / 128, 128, lig_bytes>>>(d_s0, d_s1, d_s2, d_lig, nv, nf, na, np);

	// Copy e and x from device memory to host memory.
	const size_t ex_size = (1 + nv + 1) * num_mc_tasks;
	const size_t ex_bytes = sizeof(float) * ex_size;
	h_ex.resize(ex_size);
	hipMemcpy(h_ex, d_s0, ex_bytes, hipMemcpyDeviceToHost);

	// Free device memory.
	hipFree(d_s0);
	hipFree(d_s1);
	hipFree(d_s2);
	hipFree(d_lig);
}

kernel::~kernel()
{
	for (size_t t = 0; t < sf_n; ++t)
	{
		const float* d_m = d_maps[t];
		if (d_m) hipFree(d_m);
	}
	hipFree(d_sf_d);
	hipFree(d_sf_e);
}

// -arch=sm_13 -maxrregcount=N -Xptxas=-v -ftz=true -prec-div=false -prec-sqrt=false -use_fast_math