#include "hip/hip_runtime.h"
#include <cassert>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include "cu_mc_kernel.hpp"

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
#undef  assert
#define assert(arg)
#endif

__constant__ const float* c_sf_e;
__constant__ const float* c_sf_d;
__constant__ int c_sf_ns;
__constant__ float3 c_corner0;
__constant__ float3 c_corner1;
__constant__ int3 c_num_probes;
__constant__ float c_granularity_inverse;
__constant__ const float* c_maps[sf_n];
__constant__ int c_ng;
__constant__ unsigned long c_seed;

extern __shared__ int shared[];

// TODO: Use Restrict Qualifier for Kernel Arguments
__device__  __noinline__// __forceinline__
bool evaluate(float* e, float* g, float* a, float* q, float* c, float* d, float* f, float* t, const float* x, const int nf, const int na, const int np, const float eub)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int gds = blockDim.x * gridDim.x;
	const int gd3 = 3 * gds;
	const int gd4 = 4 * gds;
	const int* const act = shared;
	const int* const beg = &act[nf];
	const int* const end = &beg[nf];
	const int* const nbr = &end[nf];
	const int* const prn = &nbr[nf];
	const float* const yy0 = (float*)&prn[nf];
	const float* const yy1 = &yy0[nf];
	const float* const yy2 = &yy1[nf];
	const float* const xy0 = &yy2[nf];
	const float* const xy1 = &xy0[nf];
	const float* const xy2 = &xy1[nf];
	const int* const brs = (int*)&xy2[nf];
	const float* const cr0 = (float*)&brs[nf - 1];
	const float* const cr1 = &cr0[na];
	const float* const cr2 = &cr1[na];
	const int* const xst = (int*)&cr2[na];
	const int* const ip0 = &xst[na];
	const int* const ip1 = &ip0[np];
	const int* const ipp = &ip1[np];

	float y, y0, y1, y2, v0, v1, v2, c0, c1, c2, e000, e100, e010, e001, a0, a1, a2, ang, sng, r0, r1, r2, r3, vs, dr, f0, f1, f2, t0, t1, t2, d0, d1, d2;
	float q0, q1, q2, q3, q00, q01, q02, q03, q11, q12, q13, q22, q23, q33, m0, m1, m2, m3, m4, m5, m6, m7, m8;
	int i, j, k, b, w, i0, i1, i2, k0, k1, k2, z;
	const float* map;

	// Apply position, orientation and torsions.
	c[i  = gid] = x[k  = gid];
	c[i += gds] = x[k += gds];
	c[i += gds] = x[k += gds];
	q[i  = gid] = x[k += gds];
	q[i += gds] = x[k += gds];
	q[i += gds] = x[k += gds];
	q[i += gds] = x[k += gds];
	y = 0.0f;
	for (k = 0, b = 0, w = 6 * gds + gid; k < nf; ++k)
	{
		// Load rotorY from memory into registers.
		y0 = c[i0  = beg[k] * gd3 + gid];
		y1 = c[i0 += gds];
		y2 = c[i0 += gds];

		// Translate orientation of active frames from quaternion into 3x3 matrix.
		if (act[k])
		{
			q0 = q[k0  = k * gd4 + gid];
			q1 = q[k0 += gds];
			q2 = q[k0 += gds];
			q3 = q[k0 += gds];
			assert(fabs(q0*q0 + q1*q1 + q2*q2 + q3*q3 - 1.0f) < 1e-3f);
			q00 = q0 * q0;
			q01 = q0 * q1;
			q02 = q0 * q2;
			q03 = q0 * q3;
			q11 = q1 * q1;
			q12 = q1 * q2;
			q13 = q1 * q3;
			q22 = q2 * q2;
			q23 = q2 * q3;
			q33 = q3 * q3;
			m0 = q00 + q11 - q22 - q33;
			m1 = 2 * (q12 - q03);
			m2 = 2 * (q02 + q13);
			m3 = 2 * (q03 + q12);
			m4 = q00 - q11 + q22 - q33;
			m5 = 2 * (q23 - q01);
			m6 = 2 * (q13 - q02);
			m7 = 2 * (q01 + q23);
			m8 = q00 - q11 - q22 + q33;
		}

		// Evaluate c and d of frame atoms. Aggregate e into y.
		for (i = beg[k], z = end[k]; i < z; ++i)
		{
			i0 = i * gd3 + gid;
			i1 = i0 + gds;
			i2 = i1 + gds;

			// The first atom of a frame is assumed to be its rotor Y.
			// TODO: avoid use of branching in short computations
			if (i == beg[k])
			{
				c0 = y0;
				c1 = y1;
				c2 = y2;
			}
			else
			{
				// Calculate coordinate from transformation matrix and offset.
				v0 = cr0[i];
				v1 = cr1[i];
				v2 = cr2[i];
				c0 = y0 + m0 * v0 + m1 * v1 + m2 * v2;
				c1 = y1 + m3 * v0 + m4 * v1 + m5 * v2;
				c2 = y2 + m6 * v0 + m7 * v1 + m8 * v2;

				// Store coordinate from registers into memory.
				c[i0] = c0;
				c[i1] = c1;
				c[i2] = c2;
			}

			// Penalize out-of-box case.
			if (c0 < c_corner0.x || c_corner1.x <= c0 || c1 < c_corner0.y || c_corner1.y <= c1 || c2 < c_corner0.z || c_corner1.z <= c2)
			{
				y += 10.0f;
				d[i0] = 0.0f;
				d[i1] = 0.0f;
				d[i2] = 0.0f;
				continue;
			}

			// Find the index of the current coordinate
			k0 = static_cast<int>((c0 - c_corner0.x) * c_granularity_inverse);
			k1 = static_cast<int>((c1 - c_corner0.y) * c_granularity_inverse);
			k2 = static_cast<int>((c2 - c_corner0.z) * c_granularity_inverse);
			assert(k0 + 1 < c_num_probes.x);
			assert(k1 + 1 < c_num_probes.y);
			assert(k2 + 1 < c_num_probes.z);
			k0 = c_num_probes.x * (c_num_probes.y * k2 + k1) + k0;

			// Retrieve the grid map and lookup the value
			map = c_maps[xst[i]];
			e000 = map[k0];
			e100 = map[k0 + 1];
			e010 = map[k0 + c_num_probes.x];
			e001 = map[k0 + c_num_probes.x * c_num_probes.y];
			y += e000;
			d[i0] = (e100 - e000) * c_granularity_inverse;
			d[i1] = (e010 - e000) * c_granularity_inverse;
			d[i2] = (e001 - e000) * c_granularity_inverse;
		}
		for (j = 0, z = nbr[k]; j < z; ++j)
		{
			i = brs[b++];
			i0 = beg[i] * gd3 + gid;
			i1 = i0 + gds;
			i2 = i1 + gds;
			c[i0] = y0 + m0 * yy0[i] + m1 * yy1[i] + m2 * yy2[i];
			c[i1] = y1 + m3 * yy0[i] + m4 * yy1[i] + m5 * yy2[i];
			c[i2] = y2 + m6 * yy0[i] + m7 * yy1[i] + m8 * yy2[i];

			// Skip inactive BRANCH frame
			if (!act[i]) continue;

			// Update a of BRANCH frame
			a0 = m0 * xy0[i] + m1 * xy1[i] + m2 * xy2[i];
			a1 = m3 * xy0[i] + m4 * xy1[i] + m5 * xy2[i];
			a2 = m6 * xy0[i] + m7 * xy1[i] + m8 * xy2[i];
			assert(fabs(a0*a0 + a1*a1 + a2*a2 - 1.0f) < 1e-3f);
			a[k0  = i * gd3 + gid] = a0;
			a[k0 += gds] = a1;
			a[k0 += gds] = a2;

			// Update q of BRANCH frame
			ang = x[w += gds] * 0.5f;
//			sng = sinf(ang);
//			r0 = cosf(ang);
			sincosf(ang, &sng, &r0);
//			sincospif(ang, &sng, &r0);
			r1 = sng * a0;
			r2 = sng * a1;
			r3 = sng * a2;
			q00 = r0 * q0 - r1 * q1 - r2 * q2 - r3 * q3;
			q01 = r0 * q1 + r1 * q0 + r2 * q3 - r3 * q2;
			q02 = r0 * q2 - r1 * q3 + r2 * q0 + r3 * q1;
			q03 = r0 * q3 + r1 * q2 - r2 * q1 + r3 * q0;
			assert(fabs(q00*q00 + q01*q01 + q02*q02 + q03*q03 - 1.0f) < 1e-3f);
			q[k0  = i * gd4 + gid] = q00;
			q[k0 += gds] = q01;
			q[k0 += gds] = q02;
			q[k0 += gds] = q03;
		}
	}
	assert(b == nf - 1);
//	assert(w == nv * gds + gid);
	assert(k == nf);

	// Calculate intra-ligand free energy.
	for (i = 0; i < np; ++i)
	{
		i0 = ip0[i] * gd3 + gid;
		i1 = i0 + gds;
		i2 = i1 + gds;
		k0 = ip1[i] * gd3 + gid;
		k1 = k0 + gds;
		k2 = k1 + gds;
		v0 = c[k0] - c[i0];
		v1 = c[k1] - c[i1];
		v2 = c[k2] - c[i2];
		vs = v0*v0 + v1*v1 + v2*v2;
		if (vs < 64.0)
		{
			j = ipp[i] + static_cast<int>(c_sf_ns * vs);
			y += c_sf_e[j];
			dr = c_sf_d[j];
			d0 = dr * v0;
			d1 = dr * v1;
			d2 = dr * v2;
			d[i0] -= d0;
			d[i1] -= d1;
			d[i2] -= d2;
			d[k0] += d0;
			d[k1] += d1;
			d[k2] += d2;
		}
	}

	// If the free energy is no better than the upper bound, refuse this conformation.
	if (y >= eub) return false;

	// Store e from register into memory.
	e[gid] = y;

	// Calculate and aggregate the force and torque of BRANCH frames to their parent frame.
	f[k0 = gid] = 0.0f;
	t[k0] = 0.0f;
	for (i = 1, z = 3 * nf; i < z; ++i)
	{
		f[k0 += gds] = 0.0f;
		t[k0] = 0.0f;
	}
//	assert(w == nv * gds + gid);
	assert(k == nf);
	while (k)
	{
		--k;

		// Load f, t and rotorY from memory into register
		k0 = k * gd3 + gid;
		k1 = k0 + gds;
		k2 = k1 + gds;
		f0 = f[k0];
		f1 = f[k1];
		f2 = f[k2];
		t0 = t[k0];
		t1 = t[k1];
		t2 = t[k2];
		y0 = c[i0  = beg[k] * gd3 + gid];
		y1 = c[i0 += gds];
		y2 = c[i0 += gds];

		// Aggregate frame atoms.
		for (i = beg[k], z = end[k]; i < z; ++i)
		{
			i0 = i * gd3 + gid;
			i1 = i0 + gds;
			i2 = i1 + gds;
			d0 = d[i0];
			d1 = d[i1];
			d2 = d[i2];

			// The derivatives with respect to the position, orientation, and torsions
			// would be the negative total force acting on the ligand,
			// the negative total torque, and the negative torque projections, respectively,
			// where the projections refer to the torque applied to the branch moved by the torsion,
			// projected on its rotation axi
			f0 += d0;
			f1 += d1;
			f2 += d2;
			if (i == beg[k]) continue;

			v0 = c[i0] - y0;
			v1 = c[i1] - y1;
			v2 = c[i2] - y2;
			t0 += v1 * d2 - v2 * d1;
			t1 += v2 * d0 - v0 * d2;
			t2 += v0 * d1 - v1 * d0;
		}

		if (k)
		{
			// Save the aggregated torque of active BRANCH frames to g.
			if (act[k])
			{
				g[w -= gds] = t0 * a[k0] + t1 * a[k1] + t2 * a[k2]; // dot product
			}

			// Aggregate the force and torque of current frame to its parent frame.
			k0 = prn[k] * gd3 + gid;
			k1 = k0 + gds;
			k2 = k1 + gds;
			f[k0] += f0;
			f[k1] += f1;
			f[k2] += f2;
			v0 = y0 - c[i0  = beg[prn[k]] * gd3 + gid];
			v1 = y1 - c[i0 += gds];
			v2 = y2 - c[i0 += gds];
			t[k0] += t0 + v1 * f2 - v2 * f1;
			t[k1] += t1 + v2 * f0 - v0 * f2;
			t[k2] += t2 + v0 * f1 - v1 * f0;
		}
	}
	assert(w == 6 * gds + gid);

	// Save the aggregated force and torque of ROOT frame to g.
	g[i0  = gid] = f0;
	g[i0 += gds] = f1;
	g[i0 += gds] = f2;
	g[i0 += gds] = t0;
	g[i0 += gds] = t1;
	g[i0 += gds] = t2;
	return true;
}

__global__
//__launch_bounds__(maxThreadsPerBlock, minBlocksPerMultiprocessor) // .maxntid nx .minnctapersm ncta
void mc(float* __restrict__ s0e, const int* __restrict__ lig, const int nv, const int nf, const int na, const int np)
{
	const int gid = blockIdx.x * blockDim.x + threadIdx.x;
	const int gds = blockDim.x * gridDim.x;
	const int nls = 5; // Number of line search trials for determining step size in BFGS
	const float eub = 40.0f * na; // A conformation will be droped if its free energy is not better than e_upper_bound.
	float* const s0x = &s0e[gds];
	float* const s0g = &s0x[(nv + 1) * gds];
	float* const s0a = &s0g[nv * gds];
	float* const s0q = &s0a[3 * nf * gds];
	float* const s0c = &s0q[4 * nf * gds];
	float* const s0d = &s0c[3 * na * gds];
	float* const s0f = &s0d[3 * na * gds];
	float* const s0t = &s0f[3 * nf * gds];
	float* const s1e = &s0t[3 * nf * gds];
	float* const s1x = &s1e[gds];
	float* const s1g = &s1x[(nv + 1) * gds];
	float* const s1a = &s1g[nv * gds];
	float* const s1q = &s1a[3 * nf * gds];
	float* const s1c = &s1q[4 * nf * gds];
	float* const s1d = &s1c[3 * na * gds];
	float* const s1f = &s1d[3 * na * gds];
	float* const s1t = &s1f[3 * nf * gds];
	float* const s2e = &s1t[3 * nf * gds];
	float* const s2x = &s2e[gds];
	float* const s2g = &s2x[(nv + 1) * gds];
	float* const s2a = &s2g[nv * gds];
	float* const s2q = &s2a[3 * nf * gds];
	float* const s2c = &s2q[4 * nf * gds];
	float* const s2d = &s2c[3 * na * gds];
	float* const s2f = &s2d[3 * na * gds];
	float* const s2t = &s2f[3 * nf * gds];
	float* const bfh = &s2t[3 * nf * gds];
	float* const bfp = &bfh[(nv*(nv+1)>>1) * gds];
	float* const bfy = &bfp[nv * gds];
	float* const bfm = &bfy[nv * gds];
	float rd0, rd1, rd2, rd3, rst;
	float sum, pg1, pga, pgc, alp, pg2, pr0, pr1, pr2, nrm, ang, sng, pq0, pq1, pq2, pq3, s1xq0, s1xq1, s1xq2, s1xq3, s2xq0, s2xq1, s2xq2, s2xq3, bpi;
	float yhy, yps, ryp, pco, bpj, bmj, ppj;
	int g, i, j, o0, o1, o2;
	hiprandState crs;

	// Load ligand into external shared memory.
	// TODO: try not caching in shared memory.
	g = 11 * nf + nf - 1 + 4 * na + 3 * np;
	o0 = threadIdx.x;
	for (i = 0, j = (g - 1) / blockDim.x; i < j; ++i)
	{
		shared[o0] = lig[o0];
		o0 += blockDim.x;
	}
	if (o0 < g)
	{
		shared[o0] = lig[o0];
	}
	__syncthreads();

	// Randomize s0x.
	hiprand_init(c_seed, gid, 0, &crs);
	rd0 = hiprand_uniform(&crs);
	s0x[o0  = gid] = rd0 * c_corner1.x + (1 - rd0) * c_corner0.x;
	rd0 = hiprand_uniform(&crs);
	s0x[o0 += gds] = rd0 * c_corner1.y + (1 - rd0) * c_corner0.y;
	rd0 = hiprand_uniform(&crs);
	s0x[o0 += gds] = rd0 * c_corner1.z + (1 - rd0) * c_corner0.z;
	rd0 = hiprand_uniform(&crs);
	rd1 = hiprand_uniform(&crs);
	rd2 = hiprand_uniform(&crs);
	rd3 = hiprand_uniform(&crs);
	rst = rsqrtf(rd0*rd0 + rd1*rd1 + rd2*rd2 + rd3*rd3);
	s0x[o0 += gds] = rd0 * rst;
	s0x[o0 += gds] = rd1 * rst;
	s0x[o0 += gds] = rd2 * rst;
	s0x[o0 += gds] = rd3 * rst;
	for (i = 6; i < nv; ++i)
	{
		s0x[o0 += gds] = hiprand_uniform(&crs);
	}
/*
	s0x[o0  = gid] =  49.799f;
	s0x[o0 += gds] = -31.025f;
	s0x[o0 += gds] =  35.312f;
	s0x[o0 += gds] = 1.0f;
	s0x[o0 += gds] = 0.0f;
	s0x[o0 += gds] = 0.0f;
	s0x[o0 += gds] = 0.0f;
	for (i = 6; i < nv; ++i)
	{
		s0x[o0 += gds] = 0.0f;
	}
*/
	evaluate(s0e, s0g, s0a, s0q, s0c, s0d, s0f, s0t, s0x, nf, na, np, eub);

	// Mutate s0x into s1x
	o0  = gid;
	s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
	o0 += gds;
	s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
	o0 += gds;
	s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
//	for (i = 3; i < nv + 1; ++i)
	for (i = 2 - nv; i < 0; ++i)
	{
		o0 += gds;
		s1x[o0] = s0x[o0];
	}
	evaluate(s1e, s1g, s1a, s1q, s1c, s1d, s1f, s1t, s1x, nf, na, np, eub);

	// Initialize the inverse Hessian matrix to identity matrix.
	// An easier option that works fine in practice is to use a scalar multiple of the identity matrix,
	// where the scaling factor is chosen to be in the range of the eigenvalues of the true Hessian.
	// See N&R for a recipe to find this initializer.
	bfh[o0 = gid] = 1.0f;
	for (j = 1; j < nv; ++j)
	{
		for (i = 0; i < j; ++i)
		{
			bfh[o0 += gds] = 0.0f;
		}
		bfh[o0 += gds] = 1.0f;
	}

	// Repeat for a number of generations.
	for (g = 0; g < c_ng; ++g)
	{
		// Use BFGS to optimize the mutated conformation s1x into local optimum s2x.
		// http://en.wikipedia.org/wiki/BFGS_method
		// http://en.wikipedia.org/wiki/Quasi-Newton_method

		// Calculate p = -h * g, where p is for descent direction, h for Hessian, and g for gradient.
		sum = bfh[o1 = gid] * s1g[o0 = gid];
		for (i = 1; i < nv; ++i)
		{
			sum += bfh[o1 += i * gds] * s1g[o0 += gds];
		}
		bfp[o2 = gid] = -sum;
		for (j = 1; j < nv; ++j)
		{
			sum = bfh[o1 = (j*(j+1)>>1) * gds + gid] * s1g[o0 = gid];
			for (i = 1; i < nv; ++i)
			{
				sum += bfh[o1 += i > j ? i * gds : gds] * s1g[o0 += gds];
			}
			bfp[o2 += gds] = -sum;
		}

		// Calculate pg = p * g = -h * g^2 < 0
		o0 = gid;
		pg1 = bfp[o0] * s1g[o0];
		for (i = 1; i < nv; ++i)
		{
			o0 += gds;
			pg1 += bfp[o0] * s1g[o0];
		}
		pga = 0.0001f * pg1;
		pgc = 0.9f * pg1;

		// Perform a line search to find an appropriate alpha.
		// Try different alpha values for nls times.
		// alpha starts with 1, and shrinks to 0.1 of itself iteration by iteration.
		alp = 1.0f;
		for (j = 0; j < nls; ++j)
		{
			// Calculate x2 = x1 + a * p.
			o0  = gid;
			s2x[o0] = s1x[o0] + alp * bfp[o0];
			o0 += gds;
			s2x[o0] = s1x[o0] + alp * bfp[o0];
			o0 += gds;
			s2x[o0] = s1x[o0] + alp * bfp[o0];
			o0 += gds;
			s1xq0 = s1x[o0];
			pr0 = bfp[o0];
			o0 += gds;
			s1xq1 = s1x[o0];
			pr1 = bfp[o0];
			o0 += gds;
			s1xq2 = s1x[o0];
			pr2 = bfp[o0];
			o0 += gds;
			s1xq3 = s1x[o0];
			assert(fabs(s1xq0*s1xq0 + s1xq1*s1xq1 + s1xq2*s1xq2 + s1xq3*s1xq3 - 1.0f) < 1e-3f);
			nrm = sqrt(pr0*pr0 + pr1*pr1 + pr2*pr2);
			ang = 0.5f * alp * nrm;
//			sng = sinf(ang) / nrm;
//			pq0 = cosf(ang);
			sincosf(ang, &sng, &pq0);
//			sincospif(ang, &sng, &pq0);
			sng /= nrm;
			pq1 = sng * pr0;
			pq2 = sng * pr1;
			pq3 = sng * pr2;
			assert(fabs(pq0*pq0 + pq1*pq1 + pq2*pq2 + pq3*pq3 - 1.0f) < 1e-3f);
			s2xq0 = pq0 * s1xq0 - pq1 * s1xq1 - pq2 * s1xq2 - pq3 * s1xq3;
			s2xq1 = pq0 * s1xq1 + pq1 * s1xq0 + pq2 * s1xq3 - pq3 * s1xq2;
			s2xq2 = pq0 * s1xq2 - pq1 * s1xq3 + pq2 * s1xq0 + pq3 * s1xq1;
			s2xq3 = pq0 * s1xq3 + pq1 * s1xq2 - pq2 * s1xq1 + pq3 * s1xq0;
			assert(fabs(s2xq0*s2xq0 + s2xq1*s2xq1 + s2xq2*s2xq2 + s2xq3*s2xq3 - 1.0f) < 1e-3f);
			s2x[o0 -= 3 * gds] = s2xq0;
			s2x[o0 += gds] = s2xq1;
			s2x[o0 += gds] = s2xq2;
			s2x[o0 += gds] = s2xq3;
			for (i = 6; i < nv; ++i)
			{
				bpi = bfp[o0];
				o0 += gds;
				s2x[o0] = s1x[o0] + alp * bpi;
			}

			// Evaluate x2, subject to Wolfe conditions http://en.wikipedia.org/wiki/Wolfe_conditions
			// 1) Armijo rule ensures that the step length alpha decreases f sufficiently.
			// 2) The curvature condition ensures that the slope has been reduced sufficiently.
			if (evaluate(s2e, s2g, s2a, s2q, s2c, s2d, s2f, s2t, s2x, nf, na, np, s1e[gid] + alp * pga))
			{
				o0 = gid;
				pg2 = bfp[o0] * s2g[o0];
				for (i = 1; i < nv; ++i)
				{
					o0 += gds;
					pg2 += bfp[o0] * s2g[o0];
				}
				if (pg2 >= pgc) break;
			}

			alp *= 0.1f;
		}

		// If no appropriate alpha can be found, restart the BFGS loop.
		if (j == nls)
		{
			// Accept x1 according to Metropolis criteria.
			if (s1e[gid] < s0e[gid])
			{
				o0 = gid;
				s0e[o0] = s1e[o0];
//				for (i = 1; i < nv + 2; ++i)
				for (i = -1 - nv; i < 0; ++i)
				{
					o0 += gds;
					s0e[o0] = s1e[o0];
				}
			}

			// Mutate s0x into s1x
			o0  = gid;
			s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
			o0 += gds;
			s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
			o0 += gds;
			s1x[o0] = s0x[o0] + hiprand_uniform(&crs);
//			for (i = 3; i < nv + 1; ++i)
			for (i = 2 - nv; i < 0; ++i)
			{
				o0 += gds;
				s1x[o0] = s0x[o0];
			}
			evaluate(s1e, s1g, s1a, s1q, s1c, s1d, s1f, s1t, s1x, nf, na, np, eub);

			// Initialize the inverse Hessian matrix to identity matrix.
			bfh[o0 = gid] = 1.0f;
			for (j = 1; j < nv; ++j)
			{
				for (i = 0; i < j; ++i)
				{
					bfh[o0 += gds] = 0.0f;
				}
				bfh[o0 += gds] = 1.0f;
			}
		}
		else
		{
			// Calculate y = g2 - g1.
			o0 = gid;
			bfy[o0] = s2g[o0] - s1g[o0];
			for (i = 1; i < nv; ++i)
			{
				o0 += gds;
				bfy[o0] = s2g[o0] - s1g[o0];
			}

			// Calculate m = -h * y.
			sum = bfh[o1 = gid] * bfy[o0 = gid];
			for (i = 1; i < nv; ++i)
			{
				sum += bfh[o1 += i * gds] * bfy[o0 += gds];
			}
			bfm[o2 = gid] = -sum;
			for (j = 1; j < nv; ++j)
			{
				sum = bfh[o1 = (j*(j+1)>>1) * gds + gid] * bfy[o0 = gid];
				for (i = 1; i < nv; ++i)
				{
					sum += bfh[o1 += i > j ? i * gds : gds] * bfy[o0 += gds];
				}
				bfm[o2 += gds] = -sum;
			}

			// Calculate yhy = -y * m = -y * (-h * y) = y * h * y.
			o0 = gid;
			yhy = -bfy[o0] * bfm[o0];
			for (i = 1; i < nv; ++i)
			{
				o0 += gds;
				yhy -= bfy[o0] * bfm[o0];
			}

			// Calculate yps = y * p.
			o0 = gid;
			yps = bfy[o0] * bfp[o0];
			for (i = 1; i < nv; ++i)
			{
				o0 += gds;
				yps += bfy[o0] * bfp[o0];
			}

			// Update Hessian matrix h.
			ryp = 1.0f / yps;
			pco = ryp * (ryp * yhy + alp);
			o2 = gid;
			for (j = 0; j < nv; ++j)
			{
				bpj = bfp[o2];
				bmj = bfm[o2];
				ppj = pco * bpj;
				bfh[o1 = (j*(j+3)>>1) * gds + gid] += (ryp * 2 * bmj + ppj) * bpj;
				for (i = j + 1; i < nv; ++i)
				{
					o0 = i * gds + gid;
					bpi = bfp[o0];
					bfh[o1 += i * gds] += ryp * (bmj * bpi + bfm[o0] * bpj) + ppj * bpi;
				}
				o2 += gds;
			}

			// Move to the next iteration, i.e. e1 = e2, x1 = x2, g1 = g2.
			o0 = gid;
			s1e[o0] = s2e[o0];
//			for (i = 1; i < 2 * (nv + 1); ++i)
			for (i = -1 - 2 * nv; i < 0; ++i)
			{
				o0 += gds;
				s1e[o0] = s2e[o0];
			}
		}
	}

	// Accept x1 according to Metropolis criteria.
	if (s1e[gid] < s0e[gid])
	{
		o0 = gid;
		s0e[o0] = s1e[o0];
//		for (i = 1; i < nv + 2; ++i)
		for (i = -1 - nv; i < 0; ++i)
		{
			o0 += gds;
			s0e[o0] = s1e[o0];
		}
	}
}

int cu_mc_kernel::initialize(const int tid, const vector<float>& h_sf_e, const vector<float>& h_sf_d, const size_t h_sf_ns, const float* h_corner0, const float* h_corner1, const int* h_num_probes, const float h_granularity_inverse, const int num_mc_tasks, const int h_ng, const unsigned long h_seed)
{
	hipSetDevice(device_id);
	this->num_mc_tasks = num_mc_tasks;

	// Initialize scoring function.
	const size_t sf_bytes = sizeof(float) * h_sf_e.size();
	checkCudaErrors(hipMalloc(&d_sf_e, sf_bytes));
	checkCudaErrors(hipMalloc(&d_sf_d, sf_bytes));
	checkCudaErrors(hipMemcpy(d_sf_e, &h_sf_e.front(), sf_bytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sf_d, &h_sf_d.front(), sf_bytes, hipMemcpyHostToDevice));
	assert(sizeof(c_sf_e)  == sizeof(d_sf_e));
	assert(sizeof(c_sf_d)  == sizeof(d_sf_d));
//	assert(sizeof(c_sf_ns) == sizeof(h_sf_ns));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_sf_e),  &d_sf_e,  sizeof(c_sf_e )));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_sf_d),  &d_sf_d,  sizeof(c_sf_d )));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_sf_ns), &h_sf_ns, sizeof(c_sf_ns)));

	// Initialize receptor.
	assert(sizeof(c_corner0) == sizeof(float) * 3);
	assert(sizeof(c_corner1) == sizeof(float) * 3);
	assert(sizeof(c_num_probes) == sizeof(int) * 3);
	assert(sizeof(c_granularity_inverse) == sizeof(h_granularity_inverse));
	assert(sizeof(c_ng) == sizeof(h_ng));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_corner0), h_corner0, sizeof(c_corner0)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_corner1), h_corner1, sizeof(c_corner1)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_num_probes), h_num_probes, sizeof(c_num_probes)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_granularity_inverse), &h_granularity_inverse, sizeof(c_granularity_inverse)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_ng), &h_ng, sizeof(c_ng)));
	assert(sizeof(d_maps) == sizeof(float*) * sf_n);
	memset(d_maps, 0, sizeof(d_maps));

	// Initialize seed.
	assert(sizeof(c_seed) == sizeof(h_seed));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_seed), &h_seed, sizeof(c_seed)));
	return 0;
}

int cu_mc_kernel::update(const int tid, const vector<vector<float> > h_maps, const vector<size_t>& xs)
{
	hipSetDevice(device_id);
	const size_t map_bytes = sizeof(float) * h_maps[xs.front()].size();
	for (int i = 0; i < xs.size(); ++i)
	{
		const size_t t = xs[i];
		float* d_m;
		checkCudaErrors(hipMalloc(&d_m, map_bytes));
		checkCudaErrors(hipMemcpy(d_m, &h_maps[t].front(), map_bytes, hipMemcpyHostToDevice));
		d_maps[t] = d_m;
	}
	assert(sizeof(c_maps) == sizeof(d_maps));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_maps), d_maps, sizeof(c_maps)));
	return 0;
}

void cu_mc_kernel::launch(vector<float>& h_ex, const vector<int>& h_lig, const int nv, const int nf, const int na, const int np)
{
	hipSetDevice(device_id);
	// Copy ligand content from host memory to device memory.
	const size_t lig_bytes = sizeof(int) * h_lig.size();
	int* d_lig;
	checkCudaErrors(hipMalloc(&d_lig, lig_bytes));
	checkCudaErrors(hipMemcpy(d_lig, &h_lig.front(), lig_bytes, hipMemcpyHostToDevice));

	// Allocate device memory for variables. 3 * (nt + 1) is sufficient for t because the torques of inactive frames are always zero.
	const size_t var_bytes = sizeof(float) * ((1 + nv + 1 + nv + 3 * nf + 4 * nf + 3 * na + 3 * na + 3 * nf + 3 * nf) * 3 + (nv * (nv + 1) >> 1) + nv * 3) * num_mc_tasks;
	float* d_s0;
	checkCudaErrors(hipMalloc(&d_s0, var_bytes));
	checkCudaErrors(hipMemset(d_s0, 0, var_bytes));

	// Invoke CUDA kernel.
	mc<<<(num_mc_tasks - 1) / 32 + 1, 32, lig_bytes>>>(d_s0, d_lig, nv, nf, na, np);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// Copy e and x from device memory to host memory.
	const size_t ex_size = (1 + nv + 1) * num_mc_tasks;
	const size_t ex_bytes = sizeof(float) * ex_size;
	h_ex.resize(ex_size);
	checkCudaErrors(hipMemcpy(&h_ex.front(), d_s0, ex_bytes, hipMemcpyDeviceToHost));

	// Free device memory.
	checkCudaErrors(hipFree(d_s0));
	checkCudaErrors(hipFree(d_lig));
}

cu_mc_kernel::~cu_mc_kernel()
{
	hipSetDevice(device_id);
	for (size_t t = 0; t < sf_n; ++t)
	{
		float* const d_m = d_maps[t];
		if (d_m) checkCudaErrors(hipFree(d_m));
	}
	checkCudaErrors(hipFree(d_sf_d));
	checkCudaErrors(hipFree(d_sf_e));
	checkCudaErrors(hipDeviceReset());
}
